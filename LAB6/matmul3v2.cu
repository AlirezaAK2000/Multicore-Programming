#include "hip/hip_runtime.h"
﻿// System includes
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <>

/**
* Matrix multiplication (CUDA Kernel) on the device: C = A * B
*/
#define TILE_WIDTH 16
#define BLOCK_SIZE 16
#define N 2048

__global__ void
matrixMulCUDA(float *C, float *A, float *B, int n)
{

	__shared__ float s_a[TILE_WIDTH][TILE_WIDTH];
	__shared__ float s_b[TILE_WIDTH][TILE_WIDTH];

	int start_row = blockDim.y * blockIdx.y + threadIdx.y * TILE_WIDTH;
	int end_row = start_row + TILE_WIDTH;
	int start_col = blockDim.x * blockIdx.x + threadIdx.x * TILE_WIDTH;
	int end_col = start_col + TILE_WIDTH;

	int tx = threadIdx.x;
	int ty = threadIdx.y;

	for (int row = start_row; row < end_row; row++)
	{
		for (int col = start_col; col < end_col; col++)
		{
			float C_val = 0;
			for (int i = 0; i < n / (TILE_WIDTH * BLOCK_SIZE); i++)
			{
				for (int j = 0; j < TILE_WIDTH; j++)
				{
					s_a[ty][tx] = A[(row * n) + (i * TILE_WIDTH * BLOCK_SIZE) + (j * TILE_WIDTH) + tx];
					s_b[ty][tx] = B[( (i * TILE_WIDTH * BLOCK_SIZE) + (j * TILE_WIDTH) + ty ) * N + col];
					
					__syncthreads();

					for(int p = 0; p < TILE_WIDTH;p++)
					{
						C_val += s_a[ty][p] * s_b[p][tx];
					}
					__syncthreads();
				}
			}
			C[row * n + col] = C_val;
		}
	}
}

void constantInit(float *data, int size, float val)
{
	for (int i = 0; i < size; ++i)
	{
		data[i] = val;
	}
}

/**
* Run a simple test of matrix multiplication using CUDA
*/
int matrixMultiply(int argc, char **argv, int n)
{
	// Allocate host memory for matrices A and B
	unsigned int size_A = n * n;
	unsigned int mem_size_A = sizeof(float) * size_A;
	float *h_A = (float *)malloc(mem_size_A);
	unsigned int size_B = n * n;
	unsigned int mem_size_B = sizeof(float) * size_B;
	float *h_B = (float *)malloc(mem_size_B);

	// Initialize host memory
	const float valB = 0.01f;
	constantInit(h_A, size_A, 1.0f);
	constantInit(h_B, size_B, valB);

	// Allocate device memory
	float *d_A, *d_B, *d_C;

	// Allocate host matrix C
	unsigned int mem_size_C = n * n * sizeof(float);
	float *h_C = (float *)malloc(mem_size_C);

	if (h_C == NULL)
	{
		fprintf(stderr, "Failed to allocate host matrix C!\n");
		exit(EXIT_FAILURE);
	}

	hipError_t error;

	error = hipMalloc((void **)&d_A, mem_size_A);

	if (error != hipSuccess)
	{
		printf("hipMalloc d_A returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void **)&d_B, mem_size_B);

	if (error != hipSuccess)
	{
		printf("hipMalloc d_B returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void **)&d_C, mem_size_C);

	if (error != hipSuccess)
	{
		printf("hipMalloc d_C returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	// copy host memory to device
	error = hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);

	if (error != hipSuccess)
	{
		printf("hipMemcpy (d_A,h_A) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice);

	if (error != hipSuccess)
	{
		printf("hipMemcpy (d_B,h_B) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	// Setup execution parameters
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE, 1);
	dim3 grid((((n - 1) / BLOCK_SIZE + 1) - 1) / TILE_WIDTH + 1, (((n - 1) / BLOCK_SIZE + 1) - 1) / TILE_WIDTH + 1, 1);

	// Create and start timer
	printf("Computing result using CUDA Kernel...\n");

	// Allocate CUDA events that we'll use for timing
	hipEvent_t start;
	error = hipEventCreate(&start);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	hipEvent_t stop;
	error = hipEventCreate(&stop);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Record the start event
	error = hipEventRecord(start, NULL);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Execute the kernel
	matrixMulCUDA<<<grid, threads>>>(d_C, d_A, d_B, n);

	error = hipGetLastError();
	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to launch kernel!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Record the stop event
	error = hipEventRecord(stop, NULL);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to record stop event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Wait for the stop event to complete
	error = hipEventSynchronize(stop);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	float msecTotal = 0.0f;
	error = hipEventElapsedTime(&msecTotal, start, stop);

	printf("Elapsed time in msec = %f\n", msecTotal);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Copy result from device to host
	error = hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost);

	if (error != hipSuccess)
	{
		printf("hipMemcpy (h_C,d_C) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	// Clean up memory
	free(h_A);
	free(h_B);
	free(h_C);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	return EXIT_SUCCESS;
}

/**
* Program main
*/
int main(int argc, char **argv)
{
	printf("[Matrix Multiply Using CUDA] - Starting...\n");

	// By default, we use device 0
	int devID = 0;
	hipSetDevice(devID);

	hipError_t error;
	hipDeviceProp_t deviceProp;
	error = hipGetDevice(&devID);

	if (error != hipSuccess)
	{
		printf("hipGetDevice returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
	}

	error = hipGetDeviceProperties(&deviceProp, devID);

	if (deviceProp.computeMode == hipComputeModeProhibited)
	{
		fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
		exit(EXIT_SUCCESS);
	}

	if (error != hipSuccess)
	{
		printf("hipGetDeviceProperties returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
	}
	else
	{
		printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
	}

	// Size of square matrices
	size_t n = N;
	// printf("[-] N = ");
	// scanf("%u", &n);

	printf("MatrixA(%d,%d), MatrixB(%d,%d)\n", n, n, n, n);

	int matrix_result = matrixMultiply(argc, argv, n);

	exit(matrix_result);
}
