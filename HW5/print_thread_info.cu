﻿
#include "hip/hip_runtime.h"


#include <stdio.h>



void print_info(unsigned int size);

__global__ void print_info_kernel()
{
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    // int p = tid * bid;
    printf("‫‪Hello‬‬ ‫‪CUDA‬‬ ‫‪I’m‬‬ ‫‪a‬‬ ‫‪thread‬‬ %d ‫‪from‬‬ ‫‬‫‪block %d \n‬‬" , tid , bid);

}

int main()
{

    print_info(100);
    
    return 0;
}

void print_info(unsigned int size)
{

    hipSetDevice(0);


    hipError_t cudaStatus;
    print_info_kernel<<<4, size>>>();
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
    }
}
