/*
* Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
*
* NOTICE TO USER:
*
* This source code is subject to NVIDIA ownership rights under U.S. and
* international Copyright laws.  Users and possessors of this source code
* are hereby granted a nonexclusive, royalty-free license to use this code
* in individual and commercial software.
*
* NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
* CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
* IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
* REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
* MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
* IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
* OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
* OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
* OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
* OR PERFORMANCE OF THIS SOURCE CODE.
*
* U.S. Government End Users.   This source code is a "commercial item" as
* that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
* "commercial computer  software"  and "commercial computer software
* documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
* and is provided to the U.S. Government only as a commercial end item.
* Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
* 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
* source code with only those rights set forth herein.
*
* Any use of this source code in individual and commercial software must
* include, in the user documentation and internal comments to the code,
* the above Disclaimer and U.S. Government End Users Notice.
*/

/* This sample queries the properties of the CUDA devices present in the system. */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

// includes, project
//#include <cutil_inline.h>

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main(int argc, char** argv)
{
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	if (deviceCount == 0)
		printf("There is no device supporting CUDA\n");
	int dev;
	for (dev = 0; dev < deviceCount; ++dev) {
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, dev);
		if (dev == 0) {
			if (deviceProp.major == 9999 && deviceProp.minor == 9999)
				printf("There is no device supporting CUDA.\n");
			else if (deviceCount == 1)
				printf("There is 1 device supporting CUDA\n");
			else
				printf("There are %d devices supporting CUDA\n", deviceCount);
		}
		printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);
		printf("  Major revision number:                         %d\n",
			deviceProp.major);
		printf("  Minor revision number:                         %d\n",
			deviceProp.minor);
		printf("  Total amount of global memory:                 %u bytes\n",
			(unsigned int)deviceProp.totalGlobalMem);
#if CUDART_VERSION >= 2000
		printf("  Number of multiprocessors:                     %d\n",
			deviceProp.multiProcessorCount);
		printf("  Number of cores:                               %d\n",
			8 * deviceProp.multiProcessorCount);
#endif
		printf("  Total amount of constant memory:               %u bytes\n",
			(unsigned int)deviceProp.totalConstMem);
		printf("  Total amount of shared memory per block:       %u bytes\n",
			(unsigned int)deviceProp.sharedMemPerBlock);
		printf("  Total number of registers available per block: %d\n",
			deviceProp.regsPerBlock);
		printf("  Warp size:                                     %d\n",
			deviceProp.warpSize);
		printf("  Maximum number of threads per block:           %d\n",
			deviceProp.maxThreadsPerBlock);
		printf("  Maximum sizes of each dimension of a block:    %d x %d x %d\n",
			deviceProp.maxThreadsDim[0],
			deviceProp.maxThreadsDim[1],
			deviceProp.maxThreadsDim[2]);
		printf("  Maximum sizes of each dimension of a grid:     %d x %d x %d\n",
			deviceProp.maxGridSize[0],
			deviceProp.maxGridSize[1],
			deviceProp.maxGridSize[2]);
		printf("  Maximum memory pitch:                          %u bytes\n",
			(unsigned int)deviceProp.memPitch);
		printf("  Texture alignment:                             %u bytes\n",
			(unsigned int)deviceProp.textureAlignment);
		printf("  Clock rate:                                    %.2f GHz\n",
			deviceProp.clockRate * 1e-6f);
#if CUDART_VERSION >= 2000
		printf("  Concurrent copy and execution:                 %s\n",
			deviceProp.deviceOverlap ? "Yes" : "No");
#endif
	}
	printf("\nTEST PASSED\n");
}
